#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "fp16.cu.h"

const int BLOCK_SIZE = 512;

__global__ void warmup(uint64_t* input_buffer, int val_num, uint64_t* block_output) {  
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    uint64_t* my_data = input_buffer + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    if (index >= val_num) return;
    uint64_t warmup_num = 0;
    for (int s = 1; s < blockDim.x; s = s * 2) {
        if (tid % (2*s) == 0) {
            warmup_num = my_data[tid] + my_data[tid + s];
        }
        __syncthreads();
    }
}

__global__ void reduceSumBase(uint64_t* input_buffer, int val_num, uint64_t* block_output) {  
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    uint64_t* my_data = input_buffer + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    if (index >= val_num) return;
    for (int s = 1; s < blockDim.x; s = s * 2) {
        if (tid % (2*s) == 0) {
            my_data[tid] += my_data[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        block_output[blockIdx.x] = my_data[0];
    }
}

__global__ void reduceSumSmallDataSet(uint64_t* input_buffer, int val_num, uint64_t* output) {
    uint64_t num = 0;
    if (threadIdx.x == 0) {
        for (int i = 0; i < val_num; i++) {
            num += *(input_buffer + i);
        }
        *output = num;
    }
}

__global__ void reduceSumShamem(uint64_t* input_buffer, int val_num, uint64_t* block_output) {  
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= val_num) return;
    __shared__ uint64_t my_data[BLOCK_SIZE];
    int tid = threadIdx.x;
    my_data[tid] = *(input_buffer + index);
    __syncthreads();
    for (int s = 1; s < blockDim.x; s = s * 2) {
        if (tid % (2*s) == 0) {
            my_data[tid] += my_data[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        block_output[blockIdx.x] = my_data[0];
    }
}

__global__ void reduceSumWarpOpt(uint64_t* input_buffer, int val_num, uint64_t* block_output) {  
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= val_num) return;
    __shared__ uint64_t my_data[BLOCK_SIZE];
    int tid = threadIdx.x;
    my_data[tid] = *(input_buffer + index);
    __syncthreads();
    for (int s = 1; s < blockDim.x; s = s * 2) {
        int idx = tid * 2 * s;
        if (idx < blockDim.x) {
            my_data[idx] += my_data[idx + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        block_output[blockIdx.x] = my_data[0];
    }
}
  
__global__ void reduceSumBankOpt(uint64_t* input_buffer, int val_num, uint64_t* block_output) {  
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= val_num) return;
    __shared__ uint64_t my_data[BLOCK_SIZE];
    int tid = threadIdx.x;
    my_data[tid] = *(input_buffer + index);
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s = s / 2) {
        if (tid < s) {
            my_data[tid] = my_data[tid] + my_data[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        block_output[blockIdx.x] = my_data[0];
    }
}

void lanuchWarmup(uint64_t* input_buffer, int val_num, int block_size, uint64_t* block_output) {
    int grid_size = (val_num + block_size - 1) / block_size;
    warmup<<<grid_size, block_size>>>(input_buffer, val_num, block_output);
    hipDeviceSynchronize();
}

void lanuchReduceSumBase(uint64_t* input_buffer, int val_num, int block_size, uint64_t* block_output) {
    int grid_size = (val_num + block_size - 1) / block_size;
    reduceSumBase<<<grid_size, block_size>>>(input_buffer, val_num, block_output);
    hipDeviceSynchronize();
}

void lanuchReduceSumBase2(uint64_t* input_buffer, int val_num, int block_size, uint64_t* block_output, const ::hipStream_t stream) {
    int grid_size = (val_num + block_size - 1) / block_size;
    reduceSumBase<<<grid_size, block_size, 0, stream>>>(input_buffer, val_num, block_output);
    int old_grid_dim = grid_size;
    grid_size = (old_grid_dim + block_size - 1) / block_size;
    reduceSumBase<<<grid_size, block_size, 0, stream>>>(block_output, old_grid_dim, input_buffer);
    reduceSumSmallDataSet<<<1, 32, 0, stream>>>(input_buffer, grid_size, block_output);
}

void lanuchReduceSumShamem(uint64_t* input_buffer, int val_num, int block_size, uint64_t* block_output, const ::hipStream_t stream = 0) {
    uint64_t shamem_size = BLOCK_SIZE * sizeof(uint64_t);
    int grid_size = (val_num + block_size - 1) / block_size;
    reduceSumShamem<<<grid_size, block_size, shamem_size, stream>>>(input_buffer, val_num, block_output);
    int old_grid_dim = grid_size;
    grid_size = (old_grid_dim + block_size - 1) / block_size;
    reduceSumShamem<<<grid_size, block_size, shamem_size, stream>>>(block_output, old_grid_dim, input_buffer);
    reduceSumSmallDataSet<<<1, 32, 0, stream>>>(input_buffer, grid_size, block_output);
}

void lanuchReduceSumWarpOpt(uint64_t* input_buffer, int val_num, int block_size, uint64_t* block_output, const ::hipStream_t stream = 0) {
    uint64_t shamem_size = BLOCK_SIZE * sizeof(uint64_t);
    int grid_size = (val_num + block_size - 1) / block_size;
    reduceSumWarpOpt<<<grid_size, block_size, shamem_size, stream>>>(input_buffer, val_num, block_output);
    int old_grid_dim = grid_size;
    grid_size = (old_grid_dim + block_size - 1) / block_size;
    reduceSumWarpOpt<<<grid_size, block_size, shamem_size, stream>>>(block_output, old_grid_dim, input_buffer);
    reduceSumSmallDataSet<<<1, 32, 0, stream>>>(input_buffer, grid_size, block_output);
}

void lanuchReduceSumBankOpt(uint64_t* input_buffer, int val_num, int block_size, uint64_t* block_output, const ::hipStream_t stream = 0) {
    uint64_t shamem_size = BLOCK_SIZE * sizeof(uint64_t);
    int grid_size = (val_num + block_size - 1) / block_size;
    reduceSumBankOpt<<<grid_size, block_size, shamem_size, stream>>>(input_buffer, val_num, block_output);
    int old_grid_dim = grid_size;
    grid_size = (old_grid_dim + block_size - 1) / block_size;
    reduceSumBankOpt<<<grid_size, block_size, shamem_size, stream>>>(block_output, old_grid_dim, input_buffer);
    reduceSumSmallDataSet<<<1, 32, 0, stream>>>(input_buffer, grid_size, block_output);
}